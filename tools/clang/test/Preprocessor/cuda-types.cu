
#include <hip/hip_runtime.h>
// Check that types, widths, etc. match on the host and device sides of CUDA
// compilations.  Note that we filter out long double, as this is intentionally
// different on host and device.

// RUN: %clang --cuda-host-only -nocudainc -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null > %T/i386-host-defines
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null > %T/i386-device-defines
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)' %T/i386-host-defines   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/i386-host-defines-filtered
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)' %T/i386-device-defines | grep -v '__LDBL\|_LONG_DOUBLE' > %T/i386-device-defines-filtered
// RUN: diff %T/i386-host-defines-filtered %T/i386-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target x86_64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null > %T/x86_64-host-defines
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target x86_64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null > %T/x86_64-device-defines
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF\|WIDTH\)' %T/x86_64-host-defines   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/x86_64-host-defines-filtered
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF\|WIDTH\)' %T/x86_64-device-defines | grep -v '__LDBL\|_LONG_DOUBLE' > %T/x86_64-device-defines-filtered
// RUN: diff %T/x86_64-host-defines-filtered %T/x86_64-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target powerpc64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null > %T/powerpc64-host-defines
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target powerpc64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null > %T/powerpc64-device-defines
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF\|WIDTH\)' %T/powerpc64-host-defines   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/powerpc64-host-defines-filtered
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF\|WIDTH\)' %T/powerpc64-device-defines | grep -v '__LDBL\|_LONG_DOUBLE' > %T/powerpc64-device-defines-filtered
// RUN: diff %T/powerpc64-host-defines-filtered %T/powerpc64-device-defines-filtered
